#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <iostream>
#include <fstream>

namespace cg = cooperative_groups;

#define CHECK(call) \
{ \
 const hipError_t error = call; \
 if (error != hipSuccess) \
 { \
 printf("Error: %s:%d, ", __FILE__, __LINE__); \
 printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
 exit(1); \
 } \
}
void initializeData(int *ip, int size)
{
    //generate different seeds for random N
    time_t tt;
    srand((unsigned int) time(&tt));
    for(int ii=0;ii<size;ii++)
    {
        ip[ii]=rand()%2;
    }
}

void printMatrix(int *C, const int nx, const int ny)
{
  int *ic=C;
  printf("\nMatrix: (%d.%d)\n",nx,ny);
  for(int iy=0;iy<ny;iy++)
  {
    for(int ix=0;ix<nx;ix++)
    {   
        if(ic[ix]==1)
          printf("%d",0);
        else
          printf(" ");
    }
    ic+=nx;
    printf("\n");
  }
  printf("\n");
}
    
 
    
void saveMatrix(int *C, char* name, const int nx, const int ny)
{
  int *ic=C;
  std::ofstream outdata; // outdata is like cin
  outdata.open(name); // opens the file
  if( !outdata ) // file couldn't be opened
  { 
    std::cerr << "Error: file could not be opened" << std::endl;
      exit(1);
  }
  for(int iy=0;iy<ny;iy++)
  {
    for(int ix=0;ix<nx;ix++)
    {
        outdata << (int) ic[ix] << " ";
    }
    ic+=nx;
    outdata <<std::endl;
  }
  outdata <<std::endl;
  outdata.close();
}


__device__ int return_element(int *Matrix, int ix, int iy, int nx, int ny)
{
    //Periodic boundry conditions
    iy=(ny+iy)%ny;
    ix=(nx+ix)%nx;
    unsigned int idx = iy*nx+ix;
    return Matrix[idx];
}
__global__ void computeGOL(int *Matrix, int* neighMatrix, int nx, int ny)
{
  unsigned int ix= threadIdx.x+blockIdx.x*blockDim.x;
  unsigned int iy = threadIdx.y+blockIdx.y*blockDim.y;
  if(ix<nx && iy<ny)
  {
      int sum= return_element(neighMatrix,ix,iy,nx,ny);
   
      if(return_element(Matrix, ix, iy, nx ,ny))
      {
          if(sum<2 || sum>3)
          {
              Matrix[iy*nx+ix]=0;
          }
      }
      else
      {
          if(sum==3)
          {
              Matrix[iy*nx+ix]=1;
          }
      }

  }
    
}
__global__ void computeNeighbours(int *Matrix,int* neighMatrix, int nx, int ny)
{
  unsigned int ix= threadIdx.x+blockIdx.x*blockDim.x;
  unsigned int iy = threadIdx.y+blockIdx.y*blockDim.y;
  if(ix<nx && iy<ny)
  {
      int sum=0;
      for(int ii=-1;ii<2;ii++)
      {
          for(int jj=-1;jj<2;jj++)
          {
              if(ii!=0 || jj!=0)
              {
                  sum+=return_element(Matrix, ix+ii, iy+jj, nx ,ny);
              }
                
          }
      }
      neighMatrix[iy*nx+ix]=sum;
  }
    
}



int main(int argc, char** argv)
{
  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));
 
  // set matrix dimension
  int nx = 200;
  int ny = 200;  

  int nxy = nx*ny;
  int nBytes = nxy * sizeof(float);
  // malloc host memory
  int *gpuRef;
  gpuRef = (int *)malloc(nBytes);
  initializeData(gpuRef,nxy);

 // set up execution configuration
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
  int NN=100000;
  // malloc device global memory
 
  int *board_Matrix;
  int *neigh_Matrix;
  clock_t begin = clock();
  hipMalloc((void **)&board_Matrix, nBytes);
  hipMalloc((void **)&neigh_Matrix, nBytes);
  // transfer data from host to device
  hipMemcpy(board_Matrix, gpuRef, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(neigh_Matrix, gpuRef, nBytes, hipMemcpyHostToDevice);
 
  //printMatrix(gpuRef,nx,ny);
 
  
  for(int ii=0; ii<NN;ii++)
  {
    computeNeighbours<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
   
    computeGOL<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
    //synchronization always occurs between kernel lunches
    //CHECK(hipDeviceSynchronize());
  }

  //synchronization is implicite for hipMemcpy
  CHECK(hipMemcpy(gpuRef, board_Matrix, nBytes, hipMemcpyDeviceToHost));
  saveMatrix(gpuRef, "Mandelbrot.txt", nx, ny);
  //printMatrix(gpuRef,nx,ny);

  // free device global memory
  hipFree(board_Matrix);
  hipFree(neigh_Matrix);
  // free host memory
  free(gpuRef);
 
  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  std::cout<<"GPU1"<<std::endl;
  std::cout<<time_spent<<std::endl;
  
  //Pinned memory
  begin = clock();
  hipHostMalloc(&gpuRef, nBytes);
  
  initializeData(gpuRef,nxy);

  // malloc device global memory
 
  hipMalloc((void **)&board_Matrix, nBytes);
  hipMalloc((void **)&neigh_Matrix, nBytes);
  // transfer data from host to device
  hipMemcpy(board_Matrix, gpuRef, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(neigh_Matrix, gpuRef, nBytes, hipMemcpyHostToDevice);

  //printMatrix(gpuRef,nx,ny);
 
  for(int ii=0; ii<NN;ii++)
  {
    computeNeighbours<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
    computeGOL<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
  }
  //synchronization is implicite for hipMemcpy
  CHECK(hipMemcpy(gpuRef, board_Matrix, nBytes, hipMemcpyDeviceToHost));
  saveMatrix(gpuRef, "Mandelbrot2.txt", nx, ny);
  //printMatrix(gpuRef,nx,ny);
  // free device global memory
  hipFree(board_Matrix);
  hipFree(neigh_Matrix);
  // free host memory
  hipHostFree(gpuRef);
  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  std::cout<<"GPU2"<<std::endl;
  std::cout<<time_spent<<std::endl;
 
   //Mapped memory
  begin = clock();
  hipHostAlloc(&board_Matrix, nBytes, hipHostMallocMapped);
  hipHostAlloc(&neigh_Matrix, nBytes, hipHostMallocMapped);
  initializeData(board_Matrix,nxy);


  //printMatrix(gpuRef,nx,ny);
 
  for(int ii=0; ii<NN;ii++)
  {
    computeNeighbours<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
    computeGOL<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
  }

  CHECK(hipDeviceSynchronize());
  saveMatrix(board_Matrix, "Mandelbrot3.txt", nx, ny);
  //printMatrix(gpuRef,nx,ny);

  // free host memory
  hipHostFree(board_Matrix);
  hipHostFree(neigh_Matrix);
  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  std::cout<<"GPU3"<<std::endl;
  std::cout<<time_spent<<std::endl;
 
    //Managed memory
  begin = clock();
  hipMallocManaged(&board_Matrix, nBytes);
  hipMallocManaged(&neigh_Matrix, nBytes);
  initializeData(board_Matrix,nxy);
  

  for(int ii=0; ii<NN;ii++)
  {
    computeNeighbours<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
    computeGOL<<<grid,block>>>(board_Matrix,neigh_Matrix, nx, ny);
  }

  CHECK(hipDeviceSynchronize());
  saveMatrix(board_Matrix, "Mandelbrot4.txt", nx, ny);
  // free host memory
  hipFree(board_Matrix);
  hipFree(neigh_Matrix);
  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  std::cout<<"GPU4"<<std::endl;
  std::cout<<time_spent<<std::endl;


  // reset device
  hipDeviceReset();


  return (0);
}